#include "hip/hip_runtime.h"
/** @file correctness.cpp
 * @brief Test the tree and tree traversal by running an instance
 * of the UnitKernel with random points and charges
 */
#include "UnitKernel.kern"
#include "ExpKernel.kern"
#include "fmmtl/KernelMatrix.hpp"
#include "fmmtl/Direct.hpp"
#include "fmmtl/numeric/random.hpp"

#include <ctime>

class timer {
public:
  timer() {
    start_time = 0;
    end_time = 0;
  }

  void tic(){
    start_time = clock();
  }

  void tic(char * c) {
    printf("%s\n",c);
    start_time = clock();
  }
  void toc() {
    end_time = clock();
    printf("Time since Tic:%f s\n", (end_time - start_time) / CLOCKS_PER_SEC);
  }

  void toc(char * c) {
    printf("%s\n", c);
    end_time = clock();
    printf("Time since Tic:%f s\n", (end_time - start_time) / CLOCKS_PER_SEC);
  }

  double start_time;
  double end_time;
};

int main(int argc, char **argv)
{
  timer tt;
  tt.tic();
  int N = 10000;  // num sources
  int M = N;  // num targets
  bool checkErrors = true;

  // Parse custom command line args
  for (int i = 1; i < argc; ++i) {
    if (strcmp(argv[i],"-N") == 0) {
      N = atoi(argv[++i]);
    } else if (strcmp(argv[i],"-M") == 0) {
      M = atoi(argv[++i]);
    } else if (strcmp(argv[i],"-nocheck") == 0) {
      checkErrors = false;
    }
  }

  // Init the FMM Kernel and options
  FMMOptions opts = get_options(argc, argv);
  //typedef UnitExpansion kernel_type;
  typedef ExpExpansion kernel_type;
  kernel_type K;

  typedef kernel_type::point_type point_type;
  typedef kernel_type::source_type source_type;
  typedef kernel_type::target_type target_type;
  typedef kernel_type::charge_type charge_type;
  typedef kernel_type::result_type result_type;

  tt.tic("make random source");
  // Init sources
  std::vector<source_type> sources = fmmtl::random_n(N);

  // Init charges
  std::vector<charge_type> charges = fmmtl::random_n(N);

  // Init targets
  std::vector<target_type> targets = fmmtl::random_n(M);
  tt.toc("end random source");
  // Build the FMM
  tt.tic("make_matrix_start");
  fmmtl::kernel_matrix<kernel_type> A{ K,targets, sources };
  tt.toc("make_matrix_end");
  A.set_options(opts);
  std::vector<result_type> result = A * charges;
  // Execute the FMM
  for (int i = 0;i < 5;i++) {
    tt.tic();
    result = A * charges;
    tt.toc();
  }
  

  tt.tic();
  // Check the result
  if (checkErrors) {
    std::cout << "Computing direct matvec..." << std::endl;

    std::vector<result_type> exact(M);

    // Compute the result with a direct matrix-vector multiplication
    fmmtl::direct(K, sources, charges, targets, exact);
    tt.toc();
    int wrong_results = 0;
    for (unsigned k = 0; k < result.size(); ++k) {
      if ((exact[k] - result[k]) / exact[k] > 1e-13) {
        std::cout << "[" << std::setw(log10(M)+1) << k << "]"
                  << " Exact: " << exact[k]
                  << ", FMM: " << result[k] << std::endl;
        std::cout << (exact[k] - result[k]) / exact[k] << std::endl;
        ++wrong_results;
      }
    }
    std::cout << "Wrong counts: " << wrong_results << " of " << M << std::endl;
  }
}
